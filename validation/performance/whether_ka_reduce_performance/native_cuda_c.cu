/*
  @ author: bcynuaa <bcynuaa@163.com>
  @ date: 2025/01/19 17:11:46
  @ license: MIT
  @ language: Julia
  @ declaration: `EtherParticlesGPU.jl` is a particle based simulation framework avialable on multi-backend GPU.
  @ description:
        # * nvcc native_cuda_c.cu -o native_cuda_c.exe -ccbin=clang 
        # * on NVIDIA GeForce RTX 4090
        # * Warm up Start Time: 6.555045 s
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define N 4096
#define N_THREADS 256
#define K_INNER_LOOP 10000000
#define K_OUTER_LOOP 10

__global__ void device_vadd(const float* a, const float* b, float* c, const int* index) {
    int I = threadIdx.x + blockIdx.x * blockDim.x;
    if (I < N) {
        int J = index[I];
        for (int step = 0; step < K_INNER_LOOP; step++) {
            c[I] += a[J] + b[J];
        }
    }
}

void shuffle(int* array, size_t n) {
    for (size_t i = 0; i < n - 1; i++) {
        size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
        int t = array[j];
        array[j] = array[i];
        array[i] = t;
    }
}

int main() {
    float* a = (float*)malloc(N * sizeof(float));
    float* b = (float*)malloc(N * sizeof(float));
    float* c = (float*)malloc(N * sizeof(float));
    int* ordered_index = (int*)malloc(N * sizeof(int));
    int* disordered_index = (int*)malloc(N * sizeof(int));
    
    for (int i = 0; i < N; i++) {
        a[i] = (float)rand() / RAND_MAX;
        b[i] = (float)rand() / RAND_MAX;
        c[i] = 0.0f;
        ordered_index[i] = i;
        disordered_index[i] = i;
    }
    shuffle(disordered_index, N);

    float* d_a, *d_b, *d_c;
    int* d_index;

    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_c, N * sizeof(float));
    hipMalloc((void**)&d_index, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_index, disordered_index, N * sizeof(int), hipMemcpyHostToDevice);

    int blocks = (N + N_THREADS - 1) / N_THREADS;
    hipDeviceSynchronize();

    printf("Warm up\n");
    device_vadd<<<blocks, N_THREADS>>>(d_a, d_b, d_c, d_index);
    hipDeviceSynchronize();

    printf("Start\n");
    // cuda event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < K_OUTER_LOOP; i++) {
        device_vadd<<<blocks, N_THREADS>>>(d_a, d_b, d_c, d_index);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f s\n", milliseconds / 1000);


    free(a);
    free(b);
    free(c);
    free(ordered_index);
    free(disordered_index);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_index);

    return 0;
}
